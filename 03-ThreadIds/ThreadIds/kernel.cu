#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void printThreadIds()
{
	printf("ThreadIdx.x = %d, ThreadIdx.y = %d, ThreadIdx.z = %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
	int nx = 16; //Total threads in X direction
	int ny = 16; //Total threads in y direction

	dim3 block(8,8);
	dim3 grid(nx/block.x, ny/block.y);

	printThreadIds << <grid, block >> > ();
	hipDeviceSynchronize();
	 
	hipDeviceReset();
	return 0;
}
