#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

/*__global__ void uniqueIndexCalcThreadidx(int* input)
{
	int tid = threadIdx.x;
	printf("ThreadIdx.x = %d, BlockIdx.x = %d, GirdDim.x = %d,   value = %d \n", tid, blockIdx.x, gridDim.x, input[tid]);
}*/

/*
* Here, we do not calculate global indices in a way that, threads with in same thread block access consecutive
* memory locations or condecutive elements in array
* 
*	0	1		2	3
*	4	5		6	7	
* 
* ArrayIndex = threadId + offset (blockId.x * blockDim.x) --> grid: M blocks * 1 block
* ArrayIndex = threadId + offset (row offset + block offset) --> grid: M blocks * N block
*			Number of threads in one row = gridDim.x * blockDim.x
*			tid + ()
*			
*/
__global__ void uniqueIndexCalcThreadidx_2x1d(int* input)
{
	int tid = threadIdx.x;
	//int offset = (blockIdx.x * blockDim.x); // grid: M blocks * 1 block
	int offset = (blockIdx.x * blockDim.x) + (blockIdx.y * blockDim.x * gridDim.x); // grid: M blocks * N block
	int gid = tid + offset;
	printf("ArrayIndex = %d, ThreadIdx.x = %d, BlockIdx.x = %d, BlockIdx.y = %d, GirdDim.x = %d, GirdDim.y = %d, value = %d \n", gid, tid, blockIdx.x, blockIdx.y, gridDim.x, gridDim.y, input[gid]);
}

/*
* Here, we do not calculate global indices in a way that, threads with in same thread block access consecutive
* memory locations or condecutive elements in array
* 
* Array Indices in grid
*	0	1		4	5
*	2	3		6	7
* 
*	8	9		12	13
*	10	11		14	15
* 
* 
* block_offset = blockDim.x * blockDim.y (number of threads in block) * blockIdx.x
* row_offset = blockDim.x * blockDim.y * girdDim.y (number of threads in a row) * blockIdx.y
* 
* 
* 
*/
__global__ void uniqueIndexCalcThreadidx_2x2d(int* input)
{
	int tid = threadIdx.x;
	
	int block_offset = blockDim.x * blockDim.y * blockIdx.x;
	int row_offset = blockDim.x * blockDim.y * gridDim.y * blockIdx.y;

	int gid = tid + block_offset + row_offset;
	printf("ArrayIndex = %d, ThreadIdx.x = %d, BlockIdx.x = %d, BlockIdx.y = %d, GirdDim.x = %d, GirdDim.y = %d, value = %d \n", gid, tid, blockIdx.x, blockIdx.y, gridDim.x, gridDim.y, input[gid]);
}

int main()
{
	//int arraySize = 8;
	int arraySize = 16;
	int arrayByteSize = sizeof(int) * arraySize;
	//int arrayData[] = {0, 5, 10, 15, 20, 25, 30, 35};
	int arrayData[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};

	printf("Array Data\n");
	for (int i = 0; i < arraySize; i++)
	{
		printf("%d ", arrayData[i]);
	}
	printf("\n \n");

	//Copying data to device from host
	int* devData; 
	hipMalloc((void**)&devData, arrayByteSize);
	hipMemcpy(devData, arrayData, arrayByteSize, hipMemcpyHostToDevice);

	/*int nx = 8; //Total threads in X direction
	int ny = 1; //Total threads in y direction

	//dim3 block(8,1); // Block size 8 * 1, grid size 1 * 1
	dim3 block(4, 1); // Block size 4 * 1, grid size 2 * 1
	dim3 grid(nx/block.x, ny/block.y);*/

	int nx = 8; //Total threads in X direction
	int ny = 2; //Total threads in y direction

	//dim3 block(8,1); // Block size 8 * 1, grid size 1 * 1
	dim3 block(4, 1); // Block size 4 * 1, grid size 2 * 1
	dim3 grid(nx / block.x, ny / block.y);

	printf("*** 2D Grid-1 : Consecutive assignment in row wise in grid ***\n\n");
	uniqueIndexCalcThreadidx_2x1d << <grid, block >> > (devData);
	hipDeviceSynchronize();
	hipDeviceReset();

	//Copying data to device from host
	int* devData2;
	hipMalloc((void**)&devData2, arrayByteSize);
	hipMemcpy(devData, arrayData, arrayByteSize, hipMemcpyHostToDevice);
	printf("*** 2D Grid-2 : Based on Memory Allocation in block ***\n\n");
	uniqueIndexCalcThreadidx_2x2d << <grid, block >> > (devData);
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}
