#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void printThreadIds()
{
	printf("ThreadIdx.x = %d, ThreadIdx.y = %d, ThreadIdx.z = %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
	printf("BlockIdx.x = %d, BlockIdx.y = %d, BlockIdx.z = %d, BlockDim.x = %d, BlockDim.y = %d, GridDim.x = %d, GridDim.y = %d \n", 
		blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
}

int main()
{
	// 256 threads, each block is 8 * 8 threads, grid is 2 * 2
	/*int nx = 16; //Total threads in X direction
	int ny = 16; //Total threads in y direction

	dim3 block(8,8);
	dim3 grid(nx/block.x, ny/block.y);*/

	//  16 Threasds each block is 2 * 2 * 2 threads, 3D grid is 2 * 2 * 2
	int nx = 4; //Total threads in X direction
	int ny = 4; //Total threads in y direction

	dim3 block(2,2);
	dim3 grid(nx/block.x, ny/block.y);

	printThreadIds << <grid, block >> > ();
	hipDeviceSynchronize();
	 
	hipDeviceReset();
	return 0;
}
