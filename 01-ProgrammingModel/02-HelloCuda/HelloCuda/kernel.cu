#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void hello_cuda()
{
	printf("Hello CUDA World !!! \n");
}

int main()
{
	int nx = 16; //Total Threads per block
	int ny = 4; // Total Blocks per grid

	dim3 block(8, 2, 1); // 4 thread in x direction, 1 in y direction, 1 in z direction
	dim3 grid(nx/block.x, ny/block.y, 1); // 8 block in x direction, 1 block in y direction, 1 block in z direction

	//Kernal launch << <grid, block>> > 
	hello_cuda << <grid, block>> > ();

	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}
