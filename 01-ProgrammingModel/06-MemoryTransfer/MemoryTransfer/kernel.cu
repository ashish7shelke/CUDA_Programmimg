#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cstring>

__global__ void memoryTransferTest(int* input)
{
	int gid = threadIdx.x + (blockIdx.x * blockDim.x);
	//if(gid < size)
	printf("gid = %d, tid = %d, value = %d \n", gid, threadIdx.x, input[gid]);
}

int main()
{
	int size = 128;
	int arraySize = size * sizeof(int);

	dim3 block(64);
	dim3 grid(2);

	int* hostInput = NULL;
	hostInput = (int*)malloc(arraySize);

	time_t clock;
	srand((unsigned)time(&clock));

	for (int i = 0; i < size; i++)
	{
		hostInput[i] = ((int)rand() & 0xff);
	}

	int* devInput;
	hipMalloc((void**)&devInput, arraySize);
	hipMemcpy(devInput, hostInput, arraySize, hipMemcpyHostToDevice);

	memoryTransferTest << < grid, block>> > (devInput);
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}
